#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <algorithm>
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpuAdd(float* gpuA, float* gpuB, float* gpuC)
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	gpuC[id] = gpuA[id] * gpuB[id];
}

int main(int argc, char** argv)
{
	if (argc < 3)
	{
		printf("Usage: %s [2^size] [tpb]\n", argv[0]);
		exit(1);
	}

	int size = 1 << atoi(argv[1]);//int atoi(const char *str) ������� atoi() ������������ ������, �� ������� ��������� �������� str, � �������� ���� int. ������ ������ ��������� ���������� ������ ������ �����. � ��������� ������ ������������ 0.
	int threads = atoi(argv[2]);
	if (size <= 0 || size >= 1000000000000000)
	{
		printf("Bad size (%d).\n", size);
		exit(1);
	}
	if (threads <= 0 || threads > 1024 || threads % 32 != 0)
	{
		printf("Bad threads-per-block (%d).\n", threads);
		exit(1);
	}

	// ������� ������� �������� ������ � ����������� ������.
	float* cpuFilled = (float*)malloc(size * sizeof(float));
	float* cpuEmpty = (float*)calloc(size, sizeof(float));
	float* cpuResult = (float*)malloc(size * sizeof(float));
	if (cpuFilled == NULL || cpuEmpty == NULL || cpuResult == NULL)
	{
		printf("malloc error.\n");
		exit(1);
	}

	// ��������� ���� ������.
	for (int i = 0; i < size; ++i)
	{
		cpuFilled[i] = float(i);
	}

	// ������� ��� �������� ������� �� ����������.
	float* gpuA;
	float* gpuB;
	float* gpuC;
	int test1 = hipMalloc((void**)&gpuA, size * sizeof(float));
	int test2 = hipMalloc((void**)&gpuB, size * sizeof(float));
	int test3 = hipMalloc((void**)&gpuC, size * sizeof(float));
	if (test1 == hipErrorOutOfMemory ||
		test2 == hipErrorOutOfMemory ||
		test3 == hipErrorOutOfMemory)
	{
		printf("hipMalloc error (%d %d %d).\n", test1, test2, test3);
		exit(1);
	}

	// ��� �� ��� - ����� ���� �� ������� �� ���������.
	// � ������� ����� ��������� ������� ���������.
	hipMemcpy(gpuA, cpuFilled, size * sizeof(float),
		hipMemcpyHostToDevice);
	hipMemcpy(gpuB, cpuFilled, size * sizeof(float),
		hipMemcpyHostToDevice);

	// ��������� ���� �� ����� ��������� �� �������� �� ���������.
	//������� ����� ������ ������ �� ������� ������ �����, ��������� � ��������� ������
	int blockCount = std::max(1, size / threads);//std::max - �������� � ������������ ����� <algorithm> - ���������� ������� �� ���� ��������.
	//printf(" %6d blocks x %4d threads: ", blockCount, threads);
	if (blockCount < 1 && blockCount > 65535)
	{
		printf("Bad block count (%d).\n", blockCount);
		exit(1);
	}
	if (threads < 1 && threads > 1024)
	{
		printf("Bad threads-per-block count (%d).\n", threads);
		exit(1);
	}

	hipMemcpy(gpuC, cpuEmpty, size * sizeof(float),
		hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	hipEventSynchronize(start);
	gpuAdd << < dim3(blockCount), dim3(threads) >> > (gpuA, gpuB, gpuC);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float elapsedTime = 0.0f;
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//hipDeviceReset();
	hipMemcpy(cpuResult, gpuC, size * sizeof(float),
		hipMemcpyDeviceToHost);

	// ����������� ������.
	hipFree(gpuA);
	hipFree(gpuB);
	hipFree(gpuC);
	free(cpuFilled);
	free(cpuEmpty);
	free(cpuResult);

	return 0;
}
