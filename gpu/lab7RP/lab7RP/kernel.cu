#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>/*Thrust is a C++ template library for CUDA based on the Standard Template Library (STL). 
Thrust allows you to implement high performance parallel applications with minimal programming effort through a high-level 
interface that is fully interoperable with CUDA C.

Thrust provides a rich collection of data parallel primitives such as scan, sort, and reduce, which can be composed 
together to implement complex algorithms with concise, readable source code. By describing your computation in terms 
of these high-level abstractions you provide Thrust with the freedom to select the most efficient implementation 
automatically. As a result, Thrust can be utilized in rapid prototyping of CUDA applications, where programmer 
productivity matters most, as well as in production, where robustness and absolute performance are crucial.

Thrust provides two vector containers, host_vector and device_vector. As the names suggest, host_vector is stored in
host memory while device_vector lives in GPU device memory. Thrust�s vector containers are just like std::vector 
in the C++ STL. Like std::vector, host_vector and device_vector are generic containers (able to store any data type) 
that can be resized dynamically.*/
#include <cstdio>
#include <cmath>

struct range_functor {
    float h;
    range_functor(float _h) :h(_h) {}
    __host__ __device__//In CUDA function type qualifiers __device__ and __host__ can be used together in which case the function is compiled for both the host and the device. This allows to eliminate copy-paste.
        float operator()(float x) {
        return h * x;
    }
};

struct sin_functor {
    __device__
        float operator()(float x) {
        return __sinf(x);//Calculate the fast approximate sine of the input argument.(single precision intrinsic functions that are only supported in device code)
    }
};

int main() {
    range_functor R(0.02);
    sin_functor Sin;
    fprintf(stderr, "%g\n", R(30.0f));
    //fprintf(stderr, "%g\n", Sin(3141592.0f/6.0f));
    thrust::host_vector<float> h1(1 << 8);//created an array of 1<<8 floats
    thrust::host_vector<float> h2(1 << 8);
    thrust::device_vector<float> d1(1 << 8);
    thrust::device_vector<float> d2(1 << 8);
    thrust::sequence(thrust::device, d1.begin(), d1.end());
    thrust::transform(d1.begin(), d1.end(), d1.begin(), R);
    thrust::transform(d1.begin(), d1.end(), d2.begin(), Sin);
    h2 = d2;
    h1 = d1;
    for (int i = 0; i < (1 << 8); i++) {
        printf("%g\t%g\n", h1[i], h2[i]);
    }
    return 0;
}