#include "hip/hip_runtime.h"
//
#include "hip/hip_runtime.h"
#include ""
//
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <time.h> 
#include <iostream>
#include <fstream>
//��������� kernel ��� device
__global__ void sum(int* a, int* b, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;//���� ���������� ������ � ����� ������� � ����� � ������������ ����� ������� � �����,
	//����������� �� ������ ����� ������� ������ N, �� return
	if (i >= N) return;
	a[i] += b[i];
}
int N = 0;//������ ��������
int* src_a, * src_b; int* dev_a, * dev_b;
LARGE_INTEGER t1, t2, f, t3, t4, t5, t6, t7, t8;
int* blocksPerGrid_gl, * threadsPerBlock_gl, * N_gl, * time_gl;
int i_gl =1;
int num = 15;
void allocateMemory(int N) {
	hipMalloc(&dev_a, sizeof(int) * N);
	hipMalloc(&dev_b, sizeof(int) * N);
	src_a = (int*)malloc(sizeof(int) * N); src_b = (int*)malloc(sizeof(int) * N);
	for (int i = 0; i < N; i++) { src_a[i] = rand(); src_b[i] = rand(); }
	hipMemcpy(dev_a, src_a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, src_b, sizeof(int) * N, hipMemcpyHostToDevice);
}
void launchKernel(int N, int threadsPerBlock, int * time_gl, int * N_gl ) {
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	printf("\nCUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	QueryPerformanceCounter(&t3);//������������ ����� ������(������ ��������) - 1024
	sum << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, N);
	hipDeviceSynchronize();
	QueryPerformanceCounter(&t4);
	double tm = double(t4.QuadPart - t3.QuadPart);// / f.QuadPart;
	std::cout << "threadsPerBlock: " << threadsPerBlock << ", time: " << tm << "\n";
	time_gl[i_gl] = tm;
	N_gl[i_gl] = N; i_gl++;
}
void testFunction() {
	int threadsPerblock_local = 1;
	std::ofstream out;          // ����� ��� ������
	out.open("C:\\Users\\stepa\\repos2\\gpu\\lab1_01\\lab1_01\\results.txt"); // �������� ���� ��� ������

	for (int i_thr = threadsPerblock_local; i_thr < 1024; i_thr *= 2)
	{
		if (out.is_open())
		{
			out << "\n\nA new test: " << std::endl;
		}
		N_gl = (int*)std::malloc(num * sizeof(int));
		time_gl = (int*)std::malloc(num * sizeof(int));
		//���  ����� ������: ...
		for (int i_N = 1 << 10; i_N <= 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
		{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
			allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
			launchKernel(i_N, i_thr, time_gl, N_gl);
		}
		std::cout << "\n\narrSize N   time , threadsPerblock_local = " << i_thr;
		out << std::endl << "arrSizeN  time , threadsPerblock_local = " << i_thr;
		for (int ipi = 1; ipi < num; ipi++)
		{
			std::cout << std::endl << N_gl[ipi] << "    " << time_gl[ipi];
			out << std::endl << N_gl[ipi] << "     " << time_gl[ipi];
		}
		free(N_gl);
		free(time_gl);
		i_gl = 1;
		
	}
}
int main() {
	hipSetDevice(0);//��������� ������ ��� ���������� ��� ������
	int size0 = 1024;//���� ����� �����?
	testFunction();
	/*std::ofstream out;          // ����� ��� ������
	out.open("C:\\Users\\stepa\\repos2\\gpu\\lab1_01\\lab1_01\\results.txt"); // �������� ���� ��� ������
	if (out.is_open())
	{
		out << "A new test: " << size0 << std::endl;
	}
	N_gl = (int*)std::malloc(num * sizeof(int));
	time_gl = (int*)std::malloc(num * sizeof(int));
	//���  ����� ������: 256
	for (int i_N = 1  << 10; i_N < 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
	{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
		allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
		launchKernel(i_N, 256, time_gl,  N_gl);
	}
	std::cout << "size N   time ";
	for (int ipi = 0; ipi < num; ipi++)
	{
		std::cout << 256 <<"    " <<   time_gl[ipi] << std::endl;
	}
	//��� ����� ������: 512
	for (int i_N = 1 << 10; i_N < 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
	{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
		allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
		launchKernel(i_N, 512, time_gl, N_gl);
	}
	std::cout << "size N   time ";
	for (int ipi = 0; ipi < num; ipi++)
	{
		std::cout << 512 << "    " << time_gl[ipi] << std::endl;
	}
	//��� ����� ������: 1024
	for (int i_N = 1 << 10; i_N < 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
	{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
		allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
		launchKernel(i_N, 1024, time_gl, N_gl);
	}
	std::cout << "size N   time ";
	for (int ipi = 0; ipi < num; ipi++)
	{
		std::cout << 1024 << "    " << time_gl[ipi] << std::endl;
	}

	//int ff = 1 << 23;
	//std::cout << "ff = " << ff;
	//int a = 16 << 1;//� ����������� ��������
	//std::cout << "\na = " << a;

	*/
}

