#include "hip/hip_runtime.h"
//
#include "hip/hip_runtime.h"
#include ""
//
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <time.h> 
#include <iostream>
#include <fstream>///��� � ���� ������? (��� �� ������ 2)
__global__ void sum(int* a, int* b, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;//���� ���������� ������ � ����� ������� � ����� � ������������ ����� ������� � �����,
	//����������� �� ������ ����� ������� ������ N, �� return
	if (i >= N) return;
	a[i] += b[i];
}
	//const char* hipGetErrorString(hipError_t error)- ���������� ��������� � ����� ������ error. 
//����� ���������� �����-�� ���� ���������...
#define CUDA_CHECK_RETURN(value) {\
hipError_t _m_cudaStat=value;\ 
if(_m_cudaStat != hipSuccess) {\
fprintf(stderr, "Error %s at line %d in file %s\n",\
hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
exit(1);\
}}

//��������� kernel ��� device

int N = 0;//������ ��������
int* src_a, * src_b; int* dev_a, * dev_b;
LARGE_INTEGER t1, t2, f, t3, t4, t5, t6, t7, t8;
int* blocksPerGrid_gl, * threadsPerBlock_gl, * N_gl;
float* time_gl;
int i_gl = 1;
int num = 15;
void allocateMemory(int N) {
	CUDA_CHECK_RETURN(hipMalloc(&dev_a, sizeof(int) * N));
	//hipMalloc(&dev_a, sizeof(int) * N);
	hipMalloc(&dev_b, sizeof(int) * N);
	src_a = (int*)malloc(sizeof(int) * N); src_b = (int*)malloc(sizeof(int) * N);
	for (int i = 0; i < N; i++) { src_a[i] = rand(); src_b[i] = rand(); }
	hipMemcpy(dev_a, src_a, sizeof(int) * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, src_b, sizeof(int) * N, hipMemcpyHostToDevice);
}
void launchKernel(int N, int threadsPerBlock, float* time_gl, int* N_gl) {
	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
	printf("\nCUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
	//QueryPerformanceCounter(&t3);//������������ ����� ������(������ ��������) - 1024
	float elapsedTime;  hipEvent_t    start, stop; // ���������� ��� ������ � ���������, ���     // �������� ����������� �����    
	hipEventCreate(&start); // �������������  
	hipEventCreate(&stop); // �������
	hipEventRecord(start, 0); // �������� (�����������) ������� start  
	sum << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, N);
	hipEventRecord(stop, 0); // �������� ������� stop  
	hipEventSynchronize(stop); // ������������� �� �������  
								//CUDA_CHECK_RETURN(hipDeviceSynchronize());  
	//CUDA_CHECK_RETURN(hipGetLastError());  
	hipEventElapsedTime(&elapsedTime, start, stop); // ���������� ������������ 
	fprintf(stderr, "gTest took %g\n", elapsedTime);
	hipEventDestroy(start); // ������������  hipEventDestroy(stop); // ������
	//sum << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, N);
	hipDeviceSynchronize();
	//QueryPerformanceCounter(&t4);
	//double tm = double(t4.QuadPart - t3.QuadPart);// / f.QuadPart;
	float tm = elapsedTime;
	std::cout << "threadsPerBlock: " << threadsPerBlock << ", time: " << tm << "\n";
	time_gl[i_gl] = tm;
	N_gl[i_gl] = N; i_gl++;
}
void testFunction() {
	int threadsPerblock_local = 1;
	std::ofstream out;          // ����� ��� ������
	out.open("..\\results.txt"); // �������� ���� ��� ������
	for (int i_thr = threadsPerblock_local; i_thr <= 1024; i_thr *= 2)//�������� �.�. �������, ������� � ���� ������������ ����� - 10
	{
		if (out.is_open())
		{
			out << "\n\n" << std::endl;
		}
		N_gl = (int*)std::malloc(num * sizeof(int));
		time_gl = (float*)std::malloc(num * sizeof(float));
		//���  ����� ������: ...
		for (int i_N = 1 << 10; i_N <= 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
		{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
			allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
			launchKernel(i_N, i_thr, time_gl, N_gl);
		}
		std::cout << "\n\narrSize N   time , threadsPerblock_local = " << i_thr;
		out << std::endl << "arrSizeN time threadsPerblock=" << i_thr;
		for (int ipi = 1; ipi < num; ipi++)
		{
			std::cout << std::endl << N_gl[ipi] << "    " << time_gl[ipi];
			out << std::endl << N_gl[ipi] << " " << time_gl[ipi];
		}
		free(N_gl);
		free(time_gl);
		i_gl = 1;

	}
}
int main() {
	setlocale(LC_ALL, "US");

	hipSetDevice(0);//��������� ������ ��� ���������� ��� ������
	int size0 = 1024;//���� ����� �����?
	//launchKernel(100000, 1025, time_gl, N_gl);
	testFunction();
	//launchKernel(1024, 10000, time_gl, N_gl);


	//out.open("..\\results.txt"); // �������� ���� ��� ������











	/*std::ofstream out;          // ����� ��� ������
	out.open("C:\\Users\\stepa\\repos2\\gpu\\lab1_01\\lab1_01\\results.txt"); // �������� ���� ��� ������
	if (out.is_open())
	{
		out << "A new test: " << size0 << std::endl;
	}
	N_gl = (int*)std::malloc(num * sizeof(int));
	time_gl = (int*)std::malloc(num * sizeof(int));
	//���  ����� ������: 256
	for (int i_N = 1  << 10; i_N < 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
	{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
		allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
		launchKernel(i_N, 256, time_gl,  N_gl);
	}
	std::cout << "size N   time ";
	for (int ipi = 0; ipi < num; ipi++)
	{
		std::cout << 256 <<"    " <<   time_gl[ipi] << std::endl;
	}
	//��� ����� ������: 512
	for (int i_N = 1 << 10; i_N < 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
	{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
		allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
		launchKernel(i_N, 512, time_gl, N_gl);
	}
	std::cout << "size N   time ";
	for (int ipi = 0; ipi < num; ipi++)
	{
		std::cout << 512 << "    " << time_gl[ipi] << std::endl;
	}
	//��� ����� ������: 1024
	for (int i_N = 1 << 10; i_N < 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
	{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
		allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
		launchKernel(i_N, 1024, time_gl, N_gl);
	}
	std::cout << "size N   time ";
	for (int ipi = 0; ipi < num; ipi++)
	{
		std::cout << 1024 << "    " << time_gl[ipi] << std::endl;
	}

	//int ff = 1 << 23;
	//std::cout << "ff = " << ff;
	//int a = 16 << 1;//� ����������� ��������
	//std::cout << "\na = " << a;

	*/
}

