#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
// ������ �����
#define BLOCK_SIZE 16
// ���, ������� ����� ����� �������� ������
#define BASE_TYPE float
// ����
// ������� ���������������� �������
__global__ void matrixTranspose(const BASE_TYPE* A,  BASE_TYPE* AT, int rows, int cols)
{
    // ������ �������� � �������� �������
    int iA = cols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
    // ������ ���������������� �������� � ����������������� �������
    int iAT = rows * (blockDim.x * blockIdx.x +
        threadIdx.x) + blockDim.y * blockIdx.y +
        threadIdx.y;
  
        AT[iAT] = A[iA];
}
// ������� ���������� �����, ������� ������ ����� � � ������� ����� b
int toMultiple(int a, int b)
{
    int mod = a % b;
    if (mod != 0)
    {
        mod = b - mod;
        return a + mod;
    }
    return a;
}
int main()
{
    // ������� �������
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // ���������� ����� � �������� �������
    int rows = 1000;
    int cols = 2000;
    // ������ ���������� ����� � �������� ������� �� �����, ������� ������� ����� (16)
    rows = toMultiple(rows, BLOCK_SIZE);
    printf("rows = %d\n", rows);
    cols = toMultiple(cols, BLOCK_SIZE);
    printf("cols = %d\n", cols);
    size_t size = rows * cols * sizeof(BASE_TYPE);
    // ��������� ������ ��� ������� �� ����� �������� �������
    BASE_TYPE* h_A = (BASE_TYPE*)malloc(size);

    // ����������������� �������
    BASE_TYPE* h_AT = (BASE_TYPE*)malloc(size);
    // ������������� �������
    for (int i = 0; i < rows * cols; ++i)
    {
        h_A[i] = rand() / (BASE_TYPE)RAND_MAX;
    }
    // ��������� ���������� ������ �� �������
    // ��� �������� �������
    BASE_TYPE* d_A = NULL;
    hipMalloc((void**)&d_A, size);
    // ��������� ���������� ������ �� ������� ��� ����������������� �������
    BASE_TYPE* d_AT = NULL;
    hipMalloc((void**)&d_AT, size);
    // �������� ������� �� CPU �� GPU
    hipMemcpy(d_A, h_A, size,
        hipMemcpyHostToDevice);
    // ���������� ������ ����� � �����
    dim3 threadsPerBlock = dim3(BLOCK_SIZE,
        BLOCK_SIZE);
    dim3 blocksPerGrid = dim3(cols / BLOCK_SIZE,
        rows / BLOCK_SIZE);
    // ������ ������ �������
    hipEventRecord(start, 0);
    // ������ ����
    matrixTranspose << <blocksPerGrid,  threadsPerBlock >> > (d_A, d_AT, rows, cols);
    // ��������� ������ ����, ��������� �������
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float KernelTime;
    hipEventElapsedTime(&KernelTime, start, stop);
    printf("KernelTime: %.2f milliseconds\n", KernelTime);
    // �������� ������� �� GPU �� CPU
    hipMemcpy(h_AT, d_AT, size,
        hipMemcpyDeviceToHost);
    // �������� ������������ ������ ����
    for (int i = 0; i < rows; i++)
        for (int j = 0; j < cols; j++)
        {
            if (h_A[i * cols + j] != h_AT[j * rows + i]) {
                fprintf(stderr, "Result verification    failed at element[% d, % d]!\n", i, j);
                exit(EXIT_FAILURE);
            }
            printf("Test PASSED\n");
            // ����������� ������ �� GPU
            hipFree(d_A);
            // ����������� ������ �� GPU
            hipFree(d_AT);
            // ����������� ������ �� CPU
            free(h_A);
            free(h_AT);
            // ������� ������� �������
            hipEventDestroy(start);
            hipEventDestroy(stop);
            printf("Done\n");
            return 0;
        }
}