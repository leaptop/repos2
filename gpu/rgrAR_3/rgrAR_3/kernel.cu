#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <thrust/sequence.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <ctime>

struct fun
{
    float alpha;
    fun(float _alpha) : alpha(_alpha)
    {}
    __host__ __device__
        float operator()(float x, float y)
    {
        return x * alpha + y;
    }
};

int main()
{
    size_t N = 1 << 20;
    size_t Nx = 1 << 10;
    size_t Ny = 1 << 10;

    clock_t start = clock();
    thrust::device_vector<float> matrix_origin_dev(N);
    thrust::device_vector<float> matrix_res_dev(N);
    thrust::sequence(matrix_origin_dev.begin(), matrix_origin_dev.end());
    thrust::counting_iterator<size_t> indices(0);
    thrust::device_vector<float> temp(Ny);
    for (size_t i = 0; i < Nx; ++i)
    {
        thrust::sequence(temp.begin(), temp.end(), 0 + i, Nx);
        thrust::copy(thrust::make_permutation_iterator(matrix_origin_dev.begin(), temp.begin()), thrust::make_permutation_iterator(matrix_origin_dev.begin(), temp.end()), matrix_res_dev.begin() + i * Ny);
    }
    thrust::host_vector<float> host_result = matrix_res_dev;
    printf("Transpose time (s) - %f\n", double(clock() - start) / CLOCKS_PER_SEC);
    //        for (int i = 0; i < Ny; ++i) {
    //            for (int j = 0; j < Nx; ++j)
                    //printf("%f\t", host_result[i * Ny + j]);
    //            printf("\n");
    //        }

    start = clock();
    fun alpha(2.25);
    thrust::device_vector<float> vectorA(N);
    thrust::device_vector<float> vectorB(N);
    thrust::sequence(vectorA.begin(), vectorA.end());
    thrust::sequence(vectorB.begin(), vectorB.end(), -1, 2);
    thrust::transform(vectorA.begin(), vectorA.end(), vectorB.begin(), vectorA.begin(), alpha);
    thrust::host_vector<float> resVector = vectorA;
    printf("SAXPY time (s) - %f\n", double(clock() - start) / CLOCKS_PER_SEC);
    //for (auto i : resVector)
    //	printf("%f\t", i);
}

