#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

const size_t N = 1 << 20;//�������
const size_t Nx = 1 << 10;//������
const size_t Ny = 1 << 10;

__global__ void transpose(float* matrixOrigin, float* matrixRes)//������������� �������
{

	size_t x = blockIdx.x * blockDim.x + threadIdx.x;
	size_t y = blockIdx.y * blockDim.y + threadIdx.y;
	size_t width = gridDim.x * blockDim.x;

	matrixRes[x + y * width] = matrixOrigin[y + x * width];
}

__global__ void saxpy(float* vectorA, float* vectorB, float alpha)//
{
	size_t index = blockIdx.x * blockDim.x + threadIdx.x;
	vectorA[index] = vectorA[index] * alpha + vectorB[index];
}

int main()
{
	clock_t start = clock();
	for (int j = 0; j < 10; ++j)
	{

		hipStream_t stream0;
		hipStreamCreate(&stream0);
		float* matrix, * matrix_dev_origin, * matrix_dev_res;
		hipHostAlloc((void**)&matrix, N * sizeof(float), hipHostMallocDefault);
		for (int i = 0; i < N; ++i)
			matrix[i] = i;

		hipMalloc((void**)&matrix_dev_origin, sizeof(float) * N);
		hipMalloc((void**)&matrix_dev_res, sizeof(float) * N);

		hipMemcpyAsync(matrix_dev_origin, matrix, sizeof(float) * N, hipMemcpyHostToDevice, stream0);
		transpose << <dim3(Nx / 32, Ny / 32), dim3(32, 32) >> > (matrix_dev_origin, matrix_dev_res);
		hipMemcpyAsync(matrix, matrix_dev_res, sizeof(float) * N, hipMemcpyDeviceToHost, stream0);
		hipStreamSynchronize(stream0);

		for (int i = 0; i < Ny; ++i)
		{
			for (int j = 0; j < Nx; ++j)
				printf("%f\t", matrix[i * Nx + j]);
			printf("\n");
		}

		hipFree(matrix_dev_origin);
		hipFree(matrix_dev_res);
		hipHostFree(matrix);
	}
	printf("transpose: ");
	printf("%d\n", clock() - start);
	start = clock();

	for (int j = 0; j < 10; ++j)
	{
		float* vecA, * vecB, * vecA_device, * vecB_device;
		hipStream_t stream0;
		hipStreamCreate(&stream0);
		hipStream_t stream1;
		hipStreamCreate(&stream1);
		hipHostAlloc((void**)&vecA, N * sizeof(float), hipHostMallocDefault);
		hipHostAlloc((void**)&vecB, N * sizeof(float), hipHostMallocDefault);

		for (int i = 0; i < N; ++i)
		{
			vecA[i] = i;
			vecB[i] = i * 2 - 1;
		}

		hipMalloc((void**)&vecA_device, sizeof(float) * N);
		hipMalloc((void**)&vecB_device, sizeof(float) * N);

		hipMemcpyAsync(vecA_device, vecA, sizeof(int) * N, hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(vecB_device, vecB, sizeof(int) * N, hipMemcpyHostToDevice, stream1);
		saxpy << <N / 2 / 1024, 1024, 0, stream0 >> > (vecA_device, vecB_device, 2.25);
		saxpy << <N / 2 / 1024, 1024, 0, stream1 >> > (vecA_device + N / 2, vecB_device + N / 2, 2.25);
		hipMemcpyAsync(vecA, vecA_device, sizeof(float) * N / 2, hipMemcpyDeviceToDevice, stream0);
		hipMemcpyAsync(vecA + N / 2, vecA_device + N / 2, sizeof(float) * N / 2, hipMemcpyDeviceToDevice, stream1);
		hipStreamSynchronize(stream0);
		hipStreamSynchronize(stream1);

		//for (int i = 0; i < N; ++i)
		//	printf("%f\t", vecA[i]);

		hipFree(vecA_device);
		hipFree(vecB_device);
		hipHostFree(vecA);
		hipHostFree(vecB);
	}
	printf("saxpy: ");
	printf("%d\n", clock() - start);
}
/*
transpose: 491044
saxpy: 89
*/