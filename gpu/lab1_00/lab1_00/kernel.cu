#include "hip/hip_runtime.h"
//
#include "hip/hip_runtime.h"
#include ""
//
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <Windows.h>
#include <time.h> 
#include <iostream>
//��������� kernel ��� device
__global__ void sum(int* a, int* b, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;//���� ���������� ������ � ����� ������� � ����� � ������������ ����� ������� � �����,
	//����������� �� ������ ����� ������� ������ N, �� return
	if (i >= N) return;
	a[i] += b[i];
}
int main() {
	//������ device
	hipSetDevice(0);//��������� ������ ��� ���������� ��� ������
	//������� ������ �� device
	int N = 10000000;//������ ��������
	int* src_a, * src_b; int* dev_a, * dev_b;
	/*
	__host____device__cudaError_t hipMalloc (void **devPtr, size_t size) Allocate memory on the device.
Parameters: devPtr - Pointer to allocated DEVICE memory; size - Requested allocation size in bytes
Returns hipSuccess, hipErrorInvalidValue, hipErrorOutOfMemory
Description: Allocates size bytes of linear memory on the device and returns in *devPtr a pointer to the allocated memory. 
The allocated memory is suitably aligned for any kind of variable. The memory is not cleared. 
hipMalloc() returns hipErrorOutOfMemory in case of failure.
Modules
www.nvidia.com CUDA Runtime API vRelease Version�|�127
The device version of hipFree cannot be used with a *devPtr allocated using the host API, and vice versa.
	*/
	hipMalloc(&dev_a, sizeof(int) * N);//������� ������ �� �������. //sizeof(int) = 4. ������ ������� ������ �� N �����(1024)
	hipMalloc(&dev_b, sizeof(int) * N);//������ �������� - ������ � ������. ������ ������ �� ���������
	src_a = (int*)malloc(sizeof(int) * N); src_b = (int*)malloc(sizeof(int) * N);
	//for (int i = 0; i < N; i++) { src_a[i] = i; src_b[i] = N - i; }
	for (int i = 0; i < N; i++) { src_a[i] = rand(); src_b[i] = rand(); }
	//��������� ������� �������� ������ ����� ������ � ��������:(� ����� ������ � ����� �� ������)
	//������ �������� - ����� ����������, ������ - ��������,
	//������ - ������ ��� ����������� � ������, ��������� - ��� ��������.
	/*
	__host__cudaError_t hipMemcpy 
	(void *dst, const void *src, size_t count, hipMemcpyKind kind)
�������� count ���� �� ������, �� ������� ��������� src � ������, �� ������� ���������
dst. kind ���������� ����������� ����������� � ����� ���� ������: hipMemcpyHostToHost, hipMemcpyHostToDevice, hipMemcpyDeviceToHost, 
hipMemcpyDeviceToDevice, or hipMemcpyDefault. ������������� ������������ hipMemcpyDefault, ��� ������� ����������� ������������ 
(�����������) �� ��������� ����������.
	*/
	/*
	__host__cudaError_t hipMemcpy (void *dst, const void *src, size_t count, hipMemcpyKind kind) Copies data between host and device.
Parameters dst - Destination memory address src - Source memory address count - Size in bytes to copy kind - Type of transfer
Modules
www.nvidia.com CUDA Runtime API vRelease Version�|�144
Returns hipSuccess, hipErrorInvalidValue, hipErrorInvalidMemcpyDirection
Description: Copies ##count## bytes from the memory area pointed to by ###src### to the memory area pointed to by ###dst###, 
where ##kind## specifies the direction of the copy, and must be one of hipMemcpyHostToHost, hipMemcpyHostToDevice, hipMemcpyDeviceToHost, 
hipMemcpyDeviceToDevice, or hipMemcpyDefault. Passing hipMemcpyDefault is recommended, in which case the type of transfer 
is inferred from the pointer values. However, hipMemcpyDefault is only allowed on systems that support unified 
virtual addressing. Calling hipMemcpy() with dst and src pointers that do not match the direction of the copy results 
in an undefined behavior.
	*/
	////���������� ������ � host �� device:
	hipMemcpy(dev_a, src_a, sizeof(int) * N, hipMemcpyHostToDevice);//������� ������
	hipMemcpy(dev_b, src_b, sizeof(int) * N, hipMemcpyHostToDevice);

	//FILETIME createTime;
	//FILETIME exitTime;
	//FILETIME kernelTime;
	//FILETIME userTime;
	//if (GetProcessTimes(GetCurrentProcess(),
	//	&createTime, &exitTime, &kernelTime, &userTime) != -1)
	//{
	//	SYSTEMTIME userSystemTime;
	//	if (FileTimeToSystemTime(&userTime, &userSystemTime) != -1)
	//		return (double)userSystemTime.wHour * 3600.0 +
	//		(double)userSystemTime.wMinute * 60.0 +
	//		(double)userSystemTime.wSecond +
	//		(double)userSystemTime.wMilliseconds / 1000.0;
	//}

	//time_t start, end;
	//time(&start);
	////getchar();//action	
	//time(&end);
	//double milliseconds = difftime(end, start);
	//printf("The time: %f seconds\n", milliseconds);

	LARGE_INTEGER t1, t2, f;
	QueryPerformanceCounter(&t1);
	int kernell_calls = 1000;
	//for (int i = 0; i < kernell_calls; i++)
	//{
		//sum << < (N + 255) / 256, 256 >> > (dev_a, dev_b, N);
		sum << < 1, N >> > (dev_a, dev_b, N);//��� ����� �������� ������ ����� ��� �������� ���� ��������? ���� � ��� ��� 
		//� ����� ������ ���������� �� ���� ��������... ���� ������ ���... ��� �������� ���� �� ������ ��� �������� �� �������-�� ���
	//}


		 // Launch the Vector Add CUDA Kernel
		int threadsPerBlock = 256;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
		sum << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, N);
		


	
	QueryPerformanceCounter(&t2);
	QueryPerformanceFrequency(&f);
	double tm = double(t2.QuadPart - t1.QuadPart);// / f.QuadPart;
	std::cout << "\ntime " << tm;

// // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);//size ����� - ��� �� N
	
	/*
	__host____device__cudaError_t hipDeviceSynchronize (void) Wait for compute device to finish.
Returns hipSuccess
Description: Blocks until the device has completed all preceding requested tasks. 
hipDeviceSynchronize() returns an error if one of the preceding tasks has failed. 
If the hipDeviceScheduleBlockingSync flag was set for this device, the host thread will block until the device has finished its work.

	*/
	hipDeviceSynchronize();
	hipMemcpy(src_a, dev_a, sizeof(int) * N, hipMemcpyDeviceToHost);
	
	
	//printf("sizeof(int) = %d", sizeof(int));
	//for (int i = 0; i < N; i++)//just to assure, that it works
	//	printf("\nsrc_a[%d] = %d", i, src_a[i]);
		
}
//
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
