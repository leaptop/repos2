#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

//#include <hip/device_functions.h>
//#include <hip/hip_runtime_api.h>
#define SH_DIM 32

__global__ void gInitializeStorage(float* storage_d, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    // ���� ��������� �������� ������������ �������� N, ������
    // ���������� ������� � �����, � ������ ������ ���������� �������
    for (int iy = j; iy < N; iy += gridDim.y * blockDim.y)
    {
        for (int ix = i; ix < N; ix += gridDim.x * blockDim.x)
        {
            storage_d[ix + iy * N] = ix + iy * N;
        }
    }
}

__global__ void gTranspose0(float* storage_d, float* storage_d_t, int N)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    for (int iy = j; iy < N; iy += gridDim.y * blockDim.y)
    {
        for (int ix = i; ix < N; ix += gridDim.x * blockDim.x)
        {
            storage_d_t[iy + ix * N] = storage_d[ix + iy * N];
        }
    }
    __syncthreads();
}

__global__ void gTranspose1(float* storage_d, float* storage_d_t, int N)
{
    __shared__ float buffer[SH_DIM][SH_DIM];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    for (int iy = j; iy < N; iy += gridDim.y * blockDim.y)
    {
        for (int ix = i; ix < N; ix += gridDim.x * blockDim.x)
        {
            buffer[threadIdx.y][threadIdx.x] = storage_d[ix + iy * N];
        }
    }
    __syncthreads();
    i = threadIdx.x + blockIdx.y * blockDim.x;
    j = threadIdx.y + blockIdx.x * blockDim.y;
    for (int iy = j; iy < N; iy += gridDim.y * blockDim.y)
    {
        for (int ix = i; ix < N; ix += gridDim.x * blockDim.x)
        {
            storage_d_t[ix + iy * N] = buffer[threadIdx.x][threadIdx.y];
        }
    }
    __syncthreads();
}

__global__ void gTranspose2(float* storage_d, float* storage_d_t, int N)
{
    __shared__ float buffer[SH_DIM][SH_DIM + 1];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    for (int iy = j; iy < N; iy += gridDim.y * blockDim.y)
    {
        for (int ix = i; ix < N; ix += gridDim.x * blockDim.x)
        {
            buffer[threadIdx.y][threadIdx.x] = storage_d[ix + iy * N];
        }
    }
    __syncthreads();
    i = threadIdx.x + blockIdx.y * blockDim.x;
    j = threadIdx.y + blockIdx.x * blockDim.y;
    for (int iy = j; iy < N; iy += gridDim.y * blockDim.y)
    {
        for (int ix = i; ix < N; ix += gridDim.x * blockDim.x)
        {
            storage_d_t[ix + iy * N] = buffer[threadIdx.x][threadIdx.y];
        }
    }
    __syncthreads();
}

void Output(float* a, int N)
{
    for (int i = 0; i < N; i += N / 4)
    {
        for (int j = 0; j < N; j += N / 4)
        {
            printf("%10.0f\t", a[j + i * N]);
        }
        printf("\n");
    }
}

int main(int argc, char* argv[])
{
    if (argc < 2)
    {
        fprintf(stderr, "USAGE: matrix <dimension>\n");
        return -1;
    }
    int N = atoi(argv[1]);
    const int max_size = 1024;
    int size = N / 32 + (N % 32 > 0);
    int dim_of_blocks = (size > max_size) ? max_size : size;
    int dim_of_threads = 32;

    float* storage_d, * storage_d_t, * storage_h;
    hipMalloc((void**)&storage_d, N * N * sizeof(float));
    hipMalloc((void**)&storage_d_t, N * N * sizeof(float));
    storage_h = (float*)calloc(N * N, sizeof(float));

    gInitializeStorage << <dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads) >> > (storage_d, N);
    hipDeviceSynchronize();
    // hipMemcpy(storage_h, storage_d, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // Output(storage_h, N);

    gTranspose0 << <dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads) >> > (storage_d, storage_d_t, N);
    hipDeviceSynchronize();
    // hipMemcpy(storage_h, storage_d_t, N*N*sizeof(float), hipMemcpyDeviceToHost);
    // Output(storage_h, N);

    gTranspose1 << <dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads) >> > (storage_d, storage_d_t, N);
    hipDeviceSynchronize();
    // hipMemcpy(storage_h, storage_d_t, N*N*sizeof(float), hipMemcpyDeviceToHost);
    // Output(storage_h, N);

    gTranspose2 << <dim3(dim_of_blocks, dim_of_blocks), dim3(dim_of_threads, dim_of_threads) >> > (storage_d, storage_d_t, N);
    hipDeviceSynchronize();
    // hipMemcpy(storage_h, storage_d_t, N * N * sizeof(float), hipMemcpyDeviceToHost);
    // Output(storage_h, N);

    hipFree(storage_d);
    hipFree(storage_d_t);
    free(storage_h);

    return 0;
}
