#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#define _USE_MATH_DEFINES
#include <math.h>
/*Thrust is a C++ template library for CUDA based on the Standard Template Library (STL).
Thrust allows you to implement high performance parallel applications with minimal programming effort through a high-level
interface that is fully interoperable with CUDA C.

Thrust provides a rich collection of data parallel primitives such as scan, sort, and reduce, which can be composed
together to implement complex algorithms with concise, readable source code. By describing your computation in terms
of these high-level abstractions you provide Thrust with the freedom to select the most efficient implementation
automatically. As a result, Thrust can be utilized in rapid prototyping of CUDA applications, where programmer
productivity matters most, as well as in production, where robustness and absolute performance are crucial.

Thrust provides two vector containers, host_vector and device_vector. As the names suggest, host_vector is stored in
host memory while device_vector lives in GPU device memory. Thrust�s vector containers are just like std::vector
in the C++ STL. Like std::vector, host_vector and device_vector are generic containers (able to store any data type)
that can be resized dynamically.*/
#include <thrust\device_vector.h>
#include <thrust\host_vector.h>
#include <thrust\functional.h>

struct func // ��������� ��� ������������� �������������� ������� �� �������
{
	float u, t, h;
	func(float _u, float _t, float _h) :
		u(_u), t(_t), h(_h) {};

	__host__ __device__//In CUDA function type qualifiers __device__ and __host__ can be used together in which case the 
		//function is compiled for both the host and the device. This allows to eliminate copy-paste.
		float operator() (const float& x, const float& y) const
	{
		return x + (y - x) * u * t / h;
	}
};

__global__ void funcCUDA(float* x, float* y, float u, float t, float h) // ����������� �������, �� ���� ��� �����
{
	int offset = threadIdx.x + blockDim.x * blockIdx.x;
	y[offset + 1] = x[offset + 1] + (x[offset] - x[offset + 1]) * u
		* t / h;
}

int main()
{
	float elapsedTime; // ������ ��������
	hipEvent_t start, stop;
	hipEventCreate(&start); // �������� ������, ������� ���
	hipEventCreate(&stop);

	hipEventRecord(start, 0); // �����
	int size = 1 << 8;
	thrust::host_vector<float> A_host(size); // ��������� ������� ��� ������ ���������� �����, ���� ����� ��� ��� � �++
	for (int i = 0; i < A_host.size(); ++i)
		A_host[i] = exp(-powf((i / 100.0 - 4.5), 2)) * 100 / (2 * sqrtf(2 * M_PI)); // ������� �����
	thrust::device_vector<float> A(size); // ���������� �������, ���� ��� ��������
	thrust::copy(A_host.begin(), A_host.end(), A.begin()); // ����������� ������� ��� � ���
	for (int i = 0; i < size; ++i)
		thrust::transform(A.begin() + 1, A.end(), A.begin(), A.begin(), func(1.1, 0.9, 1.4)); //���������� �������, ����������� ��������� ��� ��������; ����� �����, 1.1 - ���, ����� ������� ��� ����� �����, ������ � ���������������� ������, ������������ �� ���� ��� ����
	hipEventRecord(stop, 0); // ���� �������
	hipEventSynchronize(stop); // ����. ������� �������
	hipEventElapsedTime(&elapsedTime, start, stop); // ������ ������� � ���������� elapsedTime
	printf("Thrust time: ");
;	printf("%f\n", elapsedTime); // ����� �������
	// print Y
	thrust::copy(A.begin(), A.end(), std::ostream_iterator<float>(std::cout, "\n"));

	hipEventRecord(start, 0); // ����� �����
	float* vect, * vectRes, * vect_GPU, * vectRes_GPU; // ��������� �������� ���� �����, ������, ������ �� ����? ����� - ������ ��� ���������� �������� � ����� ������ ����� �������, ������ ������ � ���� ���������  ������
	vect = (float*)malloc((size) * sizeof(float)); // ������ ����������
	vectRes = (float*)malloc((size) * sizeof(float)); // ����� ������
	hipMalloc((void**)&vect_GPU, ((size) + 1) * sizeof(float)); // ������)
	hipMalloc((void**)&vectRes_GPU, ((size) + 1) * sizeof(float)); // � ����� ������
	for (int i = 0; i < (size); ++i)
		vect[i] = exp(-powf((i / 100.0 - 4.5), 2)) * 100 / (2 * sqrtf(2 * M_PI)); // ���� ���� ��������� ���� ��� ����, ������ 46, ������ ��� ��� ������ ����� ����-��� ��� �������� ����������, ������� �� ��������� ������� ��������� ������� ���� �������, ��� ������� ���� � ����� �������� ��������
	hipMemcpy(vect_GPU, vect, (size) * sizeof(float), // ���� ��� ��� - ����������� ������ �� ������ ������� �� ���� �������� �����������, � ��������� ����
		hipMemcpyHostToDevice);
	for (int i = 0; i < size; ++i) {
		funcCUDA << <1, 256 >> > (vect_GPU, vectRes_GPU, 1.1, 0.9, 1.4); // �������� ������, ��� ��������� � ������� ����
		hipMemcpy(vect_GPU, vectRes_GPU + 1, (size) * sizeof(float), hipMemcpyDeviceToDevice);
	}
	hipMemcpy(vectRes, vectRes_GPU + 1, (size) * sizeof(float),	hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0); // ���� �������
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Cuda time: ");
	printf("%f\n", elapsedTime); // ����� ����� � �� �� ����������
	for (int i = 0; i < size; i++)
	{
		printf("%f\n", vectRes[i]);
	}
	

}
// ����� - ����� ����-��� ��� ������ �������� ������� �������, ������� � ��� ��� ������ ������������, ������ � ����, + ��� �� ��� �� ����� � �����������, � 
//����� ��-���� ������ ��� ������ ����� ����� ������� ��� ��� �������, ������� �� ����� ������ ������� �� �����������.
