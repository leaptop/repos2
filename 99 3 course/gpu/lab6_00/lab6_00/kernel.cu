#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
using namespace std;

#define SIZE 1024

__global__ void kernel(int* a, int* b, int* c)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	//c[idx] = (2 * 1024 * a[idx]) / 1024 * b[idx];
	c[idx] =  a[idx] + b[idx];
}

float run(int chunk_size, int FULL_DATA_SIZE, bool use_pages, bool check_result)
{
	cout << endl << endl;
	cout << "Running on states:" << endl;
	cout << "CHUNK_SIZE     : " << chunk_size << endl;
	cout << "FULL_DATA_SIZE : " << FULL_DATA_SIZE << "; k = " << FULL_DATA_SIZE / chunk_size * 1.0f << endl;
	cout << "USE_PAGES      : " << use_pages << endl;
	cout << "CHECK_RESULT   : " << check_result << endl;
	hipDeviceProp_t prop;
	int whichDevice;
	/*hipGetDevice	(	int * 	device	 ) 	
Returns in *device the device on which the active host thread executes the device code.

Parameters:
device 	- Returns the device on which the active host thread executes the device code.
*/
	hipGetDevice(&whichDevice);
	hipGetDeviceProperties(&prop, whichDevice);

	if (!prop.deviceOverlap)
	{
		printf("Device does not support overlapping\n");//�������� �� ����������� ������� �������� � �����������(overlapping)
		return 0;
	}

	hipEvent_t start, stop;
	float elapsedTime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipStream_t stream0;/*
	hipStreamCreate	(	hipStream_t * 	pStream	 ) 	
	Creates a new asynchronous stream.
		Parameters:
	pStream - Pointer to new stream identifier*/
	hipStream_t stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	int* host_a, * host_b, * host_c;
	int* dev_a0, * dev_b0, * dev_c0;
	int* dev_a1, * dev_b1, * dev_c1;
	hipMalloc((void**)&dev_a0, chunk_size * sizeof(int));
	hipMalloc((void**)&dev_b0, chunk_size * sizeof(int));
	hipMalloc((void**)&dev_c0, chunk_size * sizeof(int));
	hipMalloc((void**)&dev_a1, chunk_size * sizeof(int));
	hipMalloc((void**)&dev_b1, chunk_size * sizeof(int));
	hipMalloc((void**)&dev_c1, chunk_size * sizeof(int));
	if (use_pages)//1.	��������� ������ �� �����
	{/*hipHostAlloc	(	void ** 	ptr, size_t 	size, unsigned int 	flags)			
Allocates size bytes of host memory that is page-locked and accessible to the device. The driver tracks the virtual 
memory ranges allocated with this function and automatically accelerates calls to functions such as hipMemcpy(). 
Since the memory can be accessed directly by the device, it can be read or written with much higher bandwidth than 
pageable memory obtained with functions such as malloc(). Allocating excessive amounts of pinned memory may degrade 
system performance, since it reduces the amount of memory available to the system for paging. As a result, this 
function is best used sparingly to allocate staging areas for data exchange between host and device.

The flags parameter enables different options to be specified that affect the allocation, as follows.

hipHostMallocDefault: This flag's value is defined to be 0 and causes hipHostAlloc() to emulate hipHostMalloc().*/
		hipHostAlloc((void**)&host_a, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
		hipHostAlloc((void**)&host_b, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
		hipHostAlloc((void**)&host_c, FULL_DATA_SIZE * sizeof(int), hipHostMallocDefault);
	}
	else
	{
		host_a = (int*)calloc(FULL_DATA_SIZE, sizeof(int));
		host_b = (int*)calloc(FULL_DATA_SIZE, sizeof(int));
		host_c = (int*)calloc(FULL_DATA_SIZE, sizeof(int));
	}

	for (int i = 0; i < FULL_DATA_SIZE; i++)
	{
		host_a[i] = 1;//�������� ������� ��� ��������
		host_b[i] = 1;
	}

	hipEventRecord(start, 0);
	for (int i = 0; i < FULL_DATA_SIZE; i += chunk_size * 2)
	{
		//cout << "     CYCLE     : " << i << endl;
		if (use_pages)//2.	����������� ������ �� ����������
		{/*hipMemcpyAsync() is asynchronous with respect to the host, so the call may return before the copy is complete. 
		 It only works on page-locked host memory and returns an error if a pointer to pageable memory is passed as input. 
		 The copy can optionally be associated to a stream by passing a non-zero stream argument. If kind is 
		 hipMemcpyHostToDevice or hipMemcpyDeviceToHost and the stream is non-zero, the copy may overlap with operations 
		 in other streams.*/
			hipMemcpyAsync(dev_a0, host_a + i, chunk_size * sizeof(int), hipMemcpyHostToDevice, stream0);//������ ������� � ������ �������
			hipMemcpyAsync(dev_a1, host_a + i + chunk_size, chunk_size * sizeof(int), hipMemcpyHostToDevice, stream1);
			hipMemcpyAsync(dev_b0, host_b + i, chunk_size * sizeof(int), hipMemcpyHostToDevice, stream0);
			hipMemcpyAsync(dev_b1, host_b + i + chunk_size, chunk_size * sizeof(int), hipMemcpyHostToDevice, stream1);
			kernel << <chunk_size / 256, 256, 0, stream0 >> > (dev_a0, dev_b0, dev_c0);//3.	���������� ���� �� ����������
			kernel << <chunk_size / 256, 256, 0, stream1 >> > (dev_a1, dev_b1, dev_c1);
			hipMemcpyAsync(host_c + i, dev_c0, chunk_size * sizeof(int), hipMemcpyDeviceToHost, stream0);//4.	����������� ������ �� ����
			hipMemcpyAsync(host_c + i + chunk_size, dev_c1, chunk_size * sizeof(int), hipMemcpyDeviceToHost, stream1);
		}
		else
		{
			hipMemcpy(dev_a0, host_a + i, chunk_size * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_a1, host_a + i + chunk_size, chunk_size * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_b0, host_b + i, chunk_size * sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(dev_b1, host_b + i + chunk_size, chunk_size * sizeof(int), hipMemcpyHostToDevice);
			kernel << <chunk_size / 256, 256, 0 >> > (dev_a0, dev_b0, dev_c0);
			kernel << <chunk_size / 256, 256, 0 >> > (dev_a1, dev_b1, dev_c1);
			hipMemcpy(host_c + i, dev_c0, chunk_size * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(host_c + i + chunk_size, dev_c1, chunk_size * sizeof(int), hipMemcpyDeviceToHost);
		}
	}
	/*hipStreamSynchronize	(	hipStream_t 	stream	 ) 	
Blocks until stream has completed all operations. If the hipDeviceScheduleBlockingSync flag was set for this device, the host thread will 
block until the stream is finished with all of its tasks.

Parameters:
stream 	- Stream identifier
*/
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipEventRecord(stop, 0);
	/*hipEventSynchronize	(	hipEvent_t 	event	 ) 	
Blocks until the event has actually been recorded. If hipEventRecord() has not been called on this event, the function returns 
hipErrorInvalidValue.*/
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "ELAPSED_TIME   : " << elapsedTime << "ms" << endl;//����� ����� ��������/���������

	if (check_result)
	{
		int sum = 0;
		for (int i = 0; i < FULL_DATA_SIZE; i++) {
			sum += host_c[i];
		}
		cout << "CHECKING_RESULT: " << sum / (FULL_DATA_SIZE * 2.0f)
			<< ((sum == FULL_DATA_SIZE * 2) ? " = SUCCESS" : " = FAIL") << endl;
	}

	hipHostFree(host_a);
	hipHostFree(host_b);
	hipHostFree(host_c);

	hipFree(dev_a0);
	hipFree(dev_b0);
	hipFree(dev_c0);
	hipFree(dev_a1);
	hipFree(dev_b1);
	hipFree(dev_c1);

	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	return elapsedTime;
}

int main()
{
	int FULL_DATA_SIZE = SIZE * SIZE * 64;
	//for (int n = SIZE; n < FULL_DATA_SIZE; n *= 2)//n - ������ ������
	//{
	//	run(n, FULL_DATA_SIZE, true, true);
	//}
	for (int i = 2; i <= 64; i*=2) {
		int N = SIZE * SIZE;
		int FULL_DATA_SIZE = SIZE * SIZE * i;
		run(N, FULL_DATA_SIZE, true, true);
		run(N, FULL_DATA_SIZE, false, true);
	}
	system("pause");
	return 0;
}