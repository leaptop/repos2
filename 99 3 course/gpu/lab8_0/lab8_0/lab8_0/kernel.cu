﻿#include "hip/hip_runtime.h"
#include ""
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <fstream>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <malloc.h>
#include <thrust\functional.h>

#define _USE_MATH_DEFINES
#include <math.h>

#define NX 4856
#define BATCH 1
#define pi 3.141592

int main() {
	hipfftHandle plan;
	hipfftComplex* data;
	hipfftComplex* data_h = (hipfftComplex*)calloc(NX * BATCH,
		sizeof(hipfftComplex));

	std::ifstream in("wolfData.txt");

	float val;
	for (int index = 0; index < NX; ++index)
	{
		in >> val >> val >> val;
		if (val != 999) {
			data_h[index].x = val;
			data_h[index].y = 0.0f;
		}
		in >> val;
	}

	hipMalloc((void**)&data, sizeof(hipfftComplex) * NX * BATCH);
	if (hipGetLastError() != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;
	}

	hipMemcpy(data, data_h, sizeof(hipfftComplex) * NX * BATCH,
		hipMemcpyHostToDevice);

	if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: Plan creation failed");
		return -1;
	}
	if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) !=
		HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
		return -1;
	}
	if (hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "Cuda error: Failed to synchronize\n");
		return -1;
	}

	hipMemcpy(data_h, data, NX * sizeof(hipfftComplex),
		hipMemcpyDeviceToHost);

	for (int i = 0; i < NX; i++)
		printf("%f\t%g\n", data_h[i].x, data_h[i].y);
	hipfftDestroy(plan);
	hipFree(data);
	free(data_h);

	return 0;
}

/*
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <malloc.h>
#include <thrust\functional.h>
#include <hipblas.h>

void print_array(float * data1,
	float * data2,
	int num_elem,
	const char * prefix) {
	printf("\n%s", prefix);
	for (int i = 0; i < num_elem; i++) {
		printf("\n%2d: %2.4f %2.4f ", i + 1, data1[i],
data2[i]);
	}
}

struct func
{
	float alpha;
	func(float _alpha) :
		alpha(_alpha) {};

	__host__ __device__
		float operator() (const float &x, const float &y) const
	{
		return x * alpha + y;
	}
};

int main() {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float time;

	/*const int num_elem = 1 << 16;
	const size_t size_in_bytes = (num_elem * sizeof(float));
	float * A_dev;
	hipMalloc((void **)&A_dev, size_in_bytes);
	float * B_dev;
	hipMalloc((void **)&B_dev, size_in_bytes);
	float * A_h;
	hipHostMalloc((void **)&A_h, size_in_bytes);
	float * B_h;
	hipHostMalloc((void **)&B_h, size_in_bytes);
	memset(A_h, 0, size_in_bytes);
	memset(B_h, 0, size_in_bytes);
	// Èíèöèàëèçàöèÿ áèáëèîòåêè CUBLAS
	hipblasHandle_t cublas_handle;
	hipblasCreate(&cublas_handle);
	for (int i = 0; i < num_elem; i++) {
		A_h[i] = (float)i;
		B_h[i] = i + 2;
	}
	//print_array(A_h, B_h, num_elem, "Before Set");
	const int num_rows = num_elem;
	const int num_cols = 1;
	const size_t elem_size = sizeof(float);

	//Êîïèðîâàíèå ìàòðèöû ñ ÷èñëîì ñòðîê num_elem è îäíèì ñòîëáöîì ñ
	//õîñòà íà óñòðîéñòâî
		hipblasSetMatrix(num_rows, num_cols, elem_size, A_h,
			num_rows, A_dev, num_rows);
	//Î÷èùàåì ìàññèâ íà óñòðîéñòâå
	hipMemcpy(B_dev, B_h, num_elem * sizeof(float),
		hipMemcpyHostToDevice);
	//hipMemset(B_dev, 0, size_in_bytes);
	// âûïîëíåíèå SingleAlphaXPlusY
	const int stride = 1;
	float alpha = 2.0F;
	hipEventRecord(start, 0);
	hipblasSaxpy(cublas_handle, num_elem, &alpha, A_dev,
		stride, B_dev, stride);
	//Êîïèðîâàíèå ìàòðèö ñ ÷èñëîì ñòðîê num_elem è îäíèì ñòîëáöîì ñ
	//óñòðîéñòâà íà õîñò
		hipblasGetMatrix(num_rows, num_cols, elem_size, A_dev,
			num_rows, A_h, num_rows);
	hipblasGetMatrix(num_rows, num_cols, elem_size, B_dev,
		num_rows, B_h, num_rows);
	//print_array(A_h, B_h, num_elem, "Intermediate Set");
	const int default_stream = 0;
	hipStreamSynchronize(default_stream);

	// Print out the arrays
	//print_array(A_h, B_h, num_elem, "After Set");
	//printf("\n");
	// Îñâîáîæäàåì ðåñóðñû íà óñòðîéñòâå
	hipblasDestroy(cublas_handle);
	hipFree(A_dev);
	hipFree(B_dev);
	// Îñâîáîæäàåì ðåñóðñû íà õîñòå
	hipHostFree(A_h);
	hipHostFree(A_h);
	hipHostFree(B_h);
	//ñáðîñ óñòðîéñòâà, ïîäãîòîâêà äëÿ âûïîëíåíèÿ íîâûõ ïðîãðàìì
	//hipDeviceReset();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("%f\n", time);


	hipEventRecord(start, 0);
	thrust::host_vector<float> A(1 << 16), B(1 << 16);
	for (int i = 0; i < A.size(); ++i)
	{
		A[i] = i;
		B[i] = i + 2;
	}
	thrust::device_vector<float> A_d(1 << 16), B_d(1 << 16);
	thrust::copy(A.begin(), A.end(), A_d.begin());
	thrust::transform(A_d.begin(), A_d.end(), B_d.begin(),
B_d.begin(), func(2));
	thrust::copy(B.begin(), B.end(), B_d.begin());
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf("%f\n", time);
}
*/
