#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <Windows.h>
#include <time.h> 
#include <iostream>
#include <fstream>

//��� �� ������ �4
#define N 3
#define M 512
__global__ void gTest() {
	__shared__ float s[N][M];
	//.....................
}

extern __shared__ float s[];
__global__ void gTest2() {
	float* a = (float*)s;
	float* b = (float*)&s[512];
	float* c = (float*)&s[1024];
	//.....................
}
//gTest2 << <100, 32, N* M * sizeof(float) >> > ();// 3-� �������� - ������ ����������� ������
//����� �������:
#include <stdio.h>
void Output(float* a, int N) {
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) 
			fprintf(stdout, "%g\t", a[j + i * N]);
		fprintf(stdout, "\n");		
	}
	fprintf(stdout, "\n\n\n");
}
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}
// ��� �� ������ �3:
__global__ void gTest1(float* a) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int I = gridDim.x * blockDim.x;
	//int J=gridDim.y*blockDim.y;    
	a[i + j * I] = (float)(threadIdx.x + blockDim.y * blockIdx.x);
}

__global__ void gTest2(float* a) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	//int I=gridDim.x*blockDim.x;  
	int J = gridDim.y * blockDim.y;
	a[j + i * J] = (float)(threadIdx.y + threadIdx.x * blockDim.y);
}
//��� �� ������ �4:
hipDeviceSetCacheConfig(hipFuncCachePreferL1);
__global__void gTest(...) {
	........
		return;
}
int main() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(gTest), hipFuncCachePreferL1);
	//.............................................
	gTest << <num_th, num_bl >> > (...);
	return 0;
}

//int n = 1000000;
//float* a;
//
//int N = 0;//������ ��������
//int* src_a, * src_b; int* dev_a, * dev_b;
//LARGE_INTEGER t1, t2, f, t3, t4, t5, t6, t7, t8;
//int* blocksPerGrid_gl, * threadsPerBlock_gl, * N_gl;
//double* time_gl;
//int i_gl = 1;
//int num = 15;
//void allocateMemory(int n) {
//	hipMalloc(&a, sizeof(float) * n);
//	a = (float*)malloc(sizeof(float) * n);
//	//src_b = (int*)malloc(sizeof(int) * N);
//	for (int i = 0; i < n; i++) { a[i] = rand(); // b[i] = rand(); 
//	}
//}
//void launchKernel(int N, int threadsPerBlock, double* time_gl, int* N_gl) {
//	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//	printf("\nCUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
//	QueryPerformanceCounter(&t3);//������������ ����� ������(������ ��������) - 1024
//	//sum << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, N);
//	gTest1 << <blocksPerGrid, threadsPerBlock >> > (a);
//	hipDeviceSynchronize();
//	QueryPerformanceCounter(&t4);
//	double tm = double((t4.QuadPart - t3.QuadPart) / (double)2600);// / f.QuadPart;// TRANSLATING TACTS TO MILLISECONDS
////	std::cout << "threadsPerBlock: " << threadsPerBlock << ", time: " << tm << "\n";
//	time_gl[i_gl] = tm;
//	N_gl[i_gl] = N; i_gl++;
//}
//void testFunction() {
//	int threadsPerblock_local = 1;
//	std::ofstream out;          // ����� ��� ������
//
//	//out.open("C:\\Users\\stepa\\repos2\\gpu\\lab1_01\\lab1_01\\results.txt"); // �������� ���� ��� ������
//	out.open("..\\results.txt"); // �������� ���� ��� ������
//
//	for (int i_thr = threadsPerblock_local; i_thr < 1024; i_thr *= 2)//�������� �.�. �������, ������� � ���� ������������ ����� - 10
//	{
//		if (out.is_open())
//		{
//			//out << "\n\n" << std::endl;
//		}
//		N_gl = (int*)std::malloc(num * sizeof(int));
//		time_gl = (double*)std::malloc(num * sizeof(double));
//		//���  ����� ������: ...
//		for (int i_N = 1 << 10; i_N <= 1 << 23; i_N *= 2)//������� �� ������������� ����� �������// �� ���� �� ������� �������� ���� �� �������� ��������
//		{//��� ���� ����� ����� ������ ������������ �����... ����� �� ����� 3
//			allocateMemory(i_N);// �� ������� ������ ������� ����� ���� �� 1<<10 �� 1<<23
//			launchKernel(i_N, i_thr, time_gl, N_gl);
//		}
//		//std::cout << "\n\narrSize N   time , threadsPerblock_local = " << i_thr;
//		//out << std::endl << "arrSizeN time threadsPerblock_local=" << i_thr;
//		//for (int ipi = 1; ipi < num; ipi++)
//		//{
//		//	std::cout << std::endl << N_gl[ipi] << "    " << time_gl[ipi];
//		//	out << std::endl << N_gl[ipi] << " " << time_gl[ipi];
//		//}
//		free(N_gl);
//		free(time_gl);
//		i_gl = 1;
//
//	}
//}
//
//int main()
//{
//	hipSetDevice(0);//��������� ������ ��� ���������� ��� ������
//	int size0 = 1024;//���� ����� �����?
//	//allocateMemory(n);
//
//	testFunction();
//
//	/*const int arraySize = 5;
//	const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	int c[arraySize] = { 0 };*/
//
//	// Add vectors in parallel.
//	/*hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addWithCuda failed!");
//		return 1;
//	}*/
//
//	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//		//c[0], c[1], c[2], c[3], c[4]);
//
//	// hipDeviceReset must be called before exiting in order for profiling and
//	// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	/*cudaStatus = hipDeviceReset();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceReset failed!");
//		return 1;
//	}*/
//
//	return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
//{
//	int* dev_a = 0;
//	int* dev_b = 0;
//	int* dev_c = 0;
//	hipError_t cudaStatus;
//
//	// Choose which GPU to run on, change this on a multi-GPU system.
//	cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	// Launch a kernel on the GPU with one thread for each element.
//	//addKernel << <1, size >> > (dev_c, dev_a, dev_b);
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	hipFree(dev_c);
//	hipFree(dev_a);
//	hipFree(dev_b);
//
//	return cudaStatus;
//}
