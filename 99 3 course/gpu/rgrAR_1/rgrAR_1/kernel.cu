
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <hipblas.h>
#include <cstdio>
#include <iostream>
#include <ctime>
#include <cstddef>

int main()
{
    size_t Nx = 1 << 10;
    size_t Ny = 1 << 10;
    size_t N = 1 << 20;
    clock_t start = clock();
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float* matrix;
    hipHostMalloc((void**)&matrix, N * sizeof(float), hipHostMallocDefault);
    for (int i = 0; i < N; ++i)
        matrix[i] = (float)i;
    float* matrix_in_dev;
    hipMalloc((void**)&matrix_in_dev, N * sizeof(float));
    float* matrix_out_dev;
    hipMalloc((void**)&matrix_out_dev, N * sizeof(float));
    hipblasSetMatrix(Ny, Nx, sizeof(float), matrix, Ny, matrix_in_dev, Ny);
    float alpha = 1.;
    float beta = 0.;
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, Nx, Ny, &alpha, matrix_in_dev, Ny, &beta, matrix_in_dev, Ny, matrix_out_dev, Nx);
    hipblasGetMatrix(Ny, Nx, sizeof(float), matrix_out_dev, Ny, matrix, Ny);
    hipStreamSynchronize(NULL);
    //        for (int i = 0; i < Ny; ++i)
    //        {
    //            for (int j = 0; j < Nx; ++j)
    //                printf("%f\t", matrix[j + i * Ny]);
    //            printf("\n");
    //        }
    hipHostFree(matrix);
    hipFree(matrix_in_dev);
    hipFree(matrix_out_dev);
    hipblasDestroy(handle);
    printf("%ld\n", (clock() - start));
    start = clock();
    hipblasHandle_t handle1;
    hipblasCreate(&handle1);
    float* vecA;
    hipHostMalloc((void**)&vecA, N * sizeof(float), hipHostMallocDefault);
    float* vecB;
    hipHostMalloc((void**)&vecB, N * sizeof(float), hipHostMallocDefault);
    for (int i = 0; i < N; ++i)
    {
        vecA[i] = (float)i;
        vecB[i] = (float)(i * 2 - 1);
    }
    float* vec_A_dev;
    hipMalloc((void**)&vec_A_dev, N * sizeof(float));
    float* vec_B_dev;
    hipMalloc((void**)&vec_B_dev, N * sizeof(float));
    hipblasSetMatrix(N, 1, sizeof(float), vecA, N, vec_A_dev, N);
    hipblasSetMatrix(N, 1, sizeof(float), vecB, N, vec_B_dev, N);
    float alpha1 = 2.25;
    hipblasSaxpy(handle1, N, &alpha1, vec_A_dev, 1, vec_B_dev, 1);
    hipblasGetMatrix(N, 1, sizeof(float), vec_B_dev, N, vecB, N);
    hipStreamSynchronize(NULL);
    //for (int i = 0; i < N; ++i)
    //	printf("%f\n", vecB[i]);
    hipblasDestroy(handle1);
    hipHostFree(vecA);
    hipHostFree(vecB);
    hipFree(vec_A_dev);
    hipFree(vec_B_dev);
    printf("%d", (clock() - start));
}
