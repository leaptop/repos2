#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib> //������� ��� malloc
#include <algorithm>
#include <iostream>

#define MAX(x, y) (((x) > (y)) ? (x) : (y))//��� �������� ��������� � ��������, ����� max ��������

#define max(a,b) \
   ({ __typeof__ (a) _a = (a); \
       __typeof__ (b) _b = (b); \
     _a > _b ? _a : _b; })

__global__//��� �� ��������� kernel - ������� ��� ���������� ���
void saxpy(int n, float a, float* x, float* y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}

int main(void)
{
    int N = 1 << 20;
    //The main function declares two pairs of arrays.
    float* x, * y, * d_x, * d_y;
    //��������� ��� � ����� ��������� �� host �������, ���������� � ������� 
    //malloc � ������� ���� 
    // � d_x, d_y ��������� �� device �������(� ������)
    x = (float*)malloc(N * sizeof(float));
    y = (float*)malloc(N * sizeof(float));

    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    saxpy << <(N + 255) / 256, 256 >> > (N, 2.0f, d_x, d_y);//��� ������ �� ��������� ��������� ������������ � �� �������� � �������� �� ������

    hipMemcpy(y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = MAX(maxError, abs(y[i] - 4.0f));
    printf("Max error: %f\n", maxError);

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}