#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>



#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M_PI 3.14159265358979323846
#define COEF 64
#define VERTCOUNT COEF* COEF * 2
#define RADIUS 10.0f
#define FGSIZE 23
#define FGSHIFT FGSIZE / 2
#define IMIN(A, B) (A < B ? A : B)
#define THREADSPERBLOCK 256
#define BLOCKSPERGRID IMIN(32, (VERTCOUNT + THREADSPERBLOCK - 1) / THREADSPERBLOCK)

typedef float (*ptr_f)(float, float, float);

struct Vertex
{
    float x, y, z;
};

__constant__ Vertex vert[VERTCOUNT];
texture<float, 3, hipReadModeElementType> df_tex;
hipArray* df_Array = 0;

float func(float x, float y, float z)
{
    return(0.5 * sqrtf(15.0 / M_PI)) * (0.5 * sqrtf(15.0 / M_PI)) * z * z * y * y * sqrtf(1.0f - z * z / RADIUS / RADIUS) / RADIUS / RADIUS / RADIUS / RADIUS;
}

float check(Vertex* v, ptr_f f)
{
    float sum = 0.0f;
    for (int i = 0; i < VERTCOUNT; ++i)
    {
        sum += f(v[i].x, v[i].y, v[i].z);
    }
    return sum;
}

void calc_f(float* arr_f, int x_size, int y_size, int z_size, ptr_f f)
{
    for (int x = 0; x < x_size; ++x)
    {
        for (int y = 0; y < y_size; ++y)
        {
            for (int z = 0; z < z_size; ++z)
            {
                arr_f[z_size * (x * y_size + y) + z] = f(x - FGSHIFT, y - FGSHIFT, z - FGSHIFT);
            }
        }
    }
}

void init_vertexes()
{
    // ���� �����, ������� �� �����
    // ���������� �� ����������� �������������� � ���������
    Vertex* temp_vert = (Vertex*)malloc(sizeof(Vertex) * VERTCOUNT);
    int i = 0;
    for (int iphi = 0; iphi < 2 * COEF; ++iphi)
    {
        for (int ipsi = 0; ipsi < COEF; ++ipsi, ++i)
        {
            float phi = iphi * M_PI / COEF;
            float psi = ipsi * M_PI / COEF;
            temp_vert[i].x = RADIUS * sinf(psi) * cosf(phi);
            temp_vert[i].y = RADIUS * sinf(psi) * sinf(phi);
            temp_vert[i].z = RADIUS * cosf(psi);
        }
    }
    printf("sumcheck = %f\n", check(temp_vert, &func) * M_PI * M_PI / COEF / COEF);
    hipMemcpyToSymbol(HIP_SYMBOL(vert), temp_vert, sizeof(Vertex) * VERTCOUNT, 0, hipMemcpyHostToDevice);
    free(temp_vert);
}

void init_texture(float* df_h)
{
    const hipExtent volumeSize = make_hipExtent(FGSIZE, FGSIZE, FGSIZE);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    hipMalloc3DArray(&df_Array, &channelDesc, volumeSize);
    hipMemcpy3DParms cpyParams = { 0 };
    cpyParams.srcPtr = make_hipPitchedPtr((void*)df_h, volumeSize.width * sizeof(float), volumeSize.width, volumeSize.height);
    cpyParams.dstArray = df_Array;
    cpyParams.extent = volumeSize;
    cpyParams.kind = hipMemcpyHostToDevice;
    hipMemcpy3D(&cpyParams);
    df_tex.normalized = false;
    df_tex.filterMode = hipFilterModeLinear;
    df_tex.addressMode[0] = hipAddressModeClamp;
    df_tex.addressMode[1] = hipAddressModeClamp;
    df_tex.addressMode[2] = hipAddressModeClamp;
    hipBindTextureToArray(df_tex, df_Array, channelDesc);
}

void release_texture()
{
    hipUnbindTexture(df_tex);
    hipFreeArray(df_Array);
}

__global__ void kernel(float* a)
{
    __shared__ float cache[THREADSPERBLOCK];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float x = vert[tid].x + FGSHIFT + 0.5f;
    float y = vert[tid].y + FGSHIFT + 0.5f;
    float z = vert[tid].z + FGSHIFT + 0.5f;
    cache[cacheIndex] = tex3D(df_tex, z, y, x);
    __syncthreads();
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (cacheIndex < s)
        {
            cache[cacheIndex] += cache[cacheIndex + s];
        }
        __syncthreads();
    }
    if (cacheIndex == 0)
    {
        a[blockIdx.x] = cache[0];
    }
}

int main(void)
{
    float* arr = (float*)malloc(sizeof(float) * FGSIZE * FGSIZE * FGSIZE);
    float* sum = (float*)malloc(sizeof(float) * BLOCKSPERGRID);
    float* sum_dev;
    hipMalloc((void**)&sum_dev, sizeof(float) * BLOCKSPERGRID);
    init_vertexes();
    calc_f(arr, FGSIZE, FGSIZE, FGSIZE, &func);
    init_texture(arr);
    kernel << <BLOCKSPERGRID, THREADSPERBLOCK >> > (sum_dev);
    hipDeviceSynchronize();
    hipMemcpy(sum, sum_dev, sizeof(float) * BLOCKSPERGRID, hipMemcpyDeviceToHost);
    float s = 0.0f;
    for (int i = 0; i < BLOCKSPERGRID; ++i)
    {
        s += sum[i];
    }
    printf("sum = %f\n", s * M_PI * M_PI / COEF / COEF);
    hipFree(sum_dev);
    free(sum);
    release_texture();
    free(arr);
    return 0;
}


/*
struct hipExtent
{
    size_t width;
    size_t height;
    size_t depth;
};

struct hipExtent make_hipExtent(size_t w, size_t h, size_t d);

struct hipChannelFormatDesc
{
    int x, y, z, w;
    enum hipChannelFormatKind f;
};

struct textureReference
{
    enum hipTextureAddressMode addressMode[3];
    struct hipChannelFormatDesc channelDesc;
    enum hipTextureFilterMode filterMode;
    int normalized;
    int sRGB;
};

hipError_t hipMalloc3DArray(struct hipArray** array, const struct hipChannelFormatDesc* desc, struct hipExtent extent, unsigned int flags = 0);

struct hipMemcpy3DParms
{
    struct hipArray* srcArray;
    struct hipPos srcPos;
    struct hipPitchedPtr srcPtr;
    struct hipArray* dstArray;
    struct hipPos dstPos;
    struct hipPitchedPtr dstPtr;
    struct hipExtent extent;
    enum hipMemcpyKind kind;
};
*/





//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
//
//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}
//
//int main()
//{
//    const int arraySize = 5;
//    const int a[arraySize] = { 1, 2, 3, 4, 5 };
//    const int b[arraySize] = { 10, 20, 30, 40, 50 };
//    int c[arraySize] = { 0 };
//
//    // Add vectors in parallel.
//    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addWithCuda failed!");
//        return 1;
//    }
//
//    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//        c[0], c[1], c[2], c[3], c[4]);
//
//    // hipDeviceReset must be called before exiting in order for profiling and
//    // tracing tools such as Nsight and Visual Profiler to show complete traces.
//    cudaStatus = hipDeviceReset();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceReset failed!");
//        return 1;
//    }
//
//    return 0;
//}
//
//// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
