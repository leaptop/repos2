#include "hip/hip_runtime.h"
// ��������� ���� ������
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

__global__ void createMatrix(int* A, const int n)
{
	// �������� ��������� ������� �� GPU
	A[threadIdx.y * n + threadIdx.x] = 10 * threadIdx.y + threadIdx.x;
}
int main()
{

	// ���-�� ����� � �������� �������
	const int n = 32;
	// ������ �������
	size_t size = n * n * sizeof(int);
	// �������� ������ ��� ������� �� CPU
	int* h_A = (int*)malloc(size);
	// �������������� �������
	for (int i = 0; i < n; i++)
		for (int j = 0; j < n; j++)
			h_A[j * n + i] = 10 * j + i;

	int* d_B = NULL;
	// �������� ������ ��� ������� �� GPU
	hipMalloc((void**)&d_B, size);

	int* d_C = NULL;
	// �������� ������ ��� ������� �� GPU
	hipMalloc((void**)&d_C, size);

	int* d_D = NULL;
	// �������� ������ ��� ������� �� GPU
	hipMalloc((void**)&d_D, size);

	// ����������� �������� ����� � ������
	dim3 threadsPerBlock = dim3(4, 25);
	dim3 threadsPerBlock2 = dim3(10, 10);
	dim3 blocksPerGrid = dim3(1);
	// ����� ����

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	// ������ ������ �������
	hipEventRecord(start, 0);
	createMatrix << <blocksPerGrid, threadsPerBlock2>> > (d_B, n);
	// ��������� ������ ����, ��������� �������
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float KernelTime;
	hipEventElapsedTime(&KernelTime, start, stop);
	printf("KernelTime: %.10f milliseconds\n", KernelTime);


	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventCreate(&stop2);
	hipEventRecord(start2, 0);
	createMatrix << <blocksPerGrid, threadsPerBlock >> > (d_C, n);
	// ��������� ������ ����, ��������� �������
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	float KernelTime2;
	hipEventElapsedTime(&KernelTime2, start2, stop2);
	printf("KernelTime: %.10f milliseconds\n", KernelTime2);

	hipEvent_t start3, stop3;
	hipEventCreate(&start3);
	hipEventCreate(&stop3);
	hipEventRecord(start3, 0);
	createMatrix << <blocksPerGrid, threadsPerBlock2 >> > (d_D, n);
	// ��������� ������ ����, ��������� �������
	hipEventRecord(stop3, 0);
	hipEventSynchronize(stop3);
	float KernelTime3;
	hipEventElapsedTime(&KernelTime3, start3, stop3);
	printf("KernelTime: %.10f milliseconds\n", KernelTime3);


	//// �������� ������ ��� ������� B, ����� ����������� �� GPU �� CPU
	//int* h_B = (int*)malloc(size);
	//// �������� ������� �� GPU �� CPU
	//hipMemcpy(h_B, d_B, size,
	//	hipMemcpyDeviceToHost);
	//// ��������� ���������� ������� � � ������� �
	//for (int i = 0; i < n; i++)
	//	for (int j = 0; j < n; j++)
	//		if (h_A[j * n + i] != h_B[j * n + i]) {
	//			printf("h_A[%d] != h_B[%d]\n", j * n
	//				+ i, j * n + i);
	//		}
	// ����������� ������ �� GPU
	hipFree(d_B);
	// ����������� ������ �� CPU
	free(h_A);
	//free(h_B);
	return 0;
}